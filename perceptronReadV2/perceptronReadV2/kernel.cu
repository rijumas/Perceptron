#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>

using namespace std;

const string training_image_fn = "t10k-images.idx3-ubyte";
const string training_label_fn = "t10k-labels.idx1-ubyte";

const int nTraining = 10000;
const int height = 28;
const int width = 28;
const int neuronas = 10;

ifstream image;
ifstream label;

vector<double> input((height* width) + 1);
vector<double> weights((height* width + 1)* neuronas, 0.0);
int errors = 0;

__global__ void addKernel(const double* d_input, const double* d_weights, double* d_y, int height, int width) {
    int neuron = blockIdx.x;
    int i = threadIdx.x;

    __shared__ double sharedSum[785];
    sharedSum[i] = d_input[i] * d_weights[neuron * (height * width + 1) + i];
    __syncthreads();

    // Reduce sum within the block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (i < stride) {
            sharedSum[i] += sharedSum[i + stride];
        }
        __syncthreads();
    }

    if (i == 0) {
        d_y[neuron] = sharedSum[0];
        if (d_y[neuron] > 0.0) {
            d_y[neuron] = 1.0;
        }
        else {
            d_y[neuron] = 0.0;
        }
    }
}

int main() {
    std::string filename = "weights.txt";
    std::ifstream inFile(filename);
    if (!inFile) {
        cerr << "No se pudo abrir el archivo para leer." << std::endl;
    }
    else {
        for (size_t i = 0; i < weights.size(); ++i) {
            inFile >> weights[i];
        }
        inFile.close();
    }

    image.open(training_image_fn.c_str(), ios::in | ios::binary);
    label.open(training_label_fn.c_str(), ios::in | ios::binary);

    char number;
    for (int i = 1; i <= 16; ++i) {
        image.read(&number, sizeof(char));
    }
    for (int i = 1; i <= 8; ++i) {
        label.read(&number, sizeof(char));
    }

    //int cantidad = 1;
    //while (cantidad > 0) {
        for (int k = 0; k < nTraining; k++) {
            vector<double> labels(neuronas, 0.0);
            char n;
            for (int i = 0; i < ((height * width) + 1); i++) {
                if (i == 0) {
                    input[0] = 1.0;
                }
                else {
                    image.read(&n, sizeof(char));
                    if (n == 0)
                        input[i] = 0.0;
                    else
                        input[i] = 1.0;
                }
            }
            label.read(&n, sizeof(char));
            labels[n] = 1.0;

            vector<double> y(neuronas, 0.0);
            double* d_weights;
            double* d_input;
            double* d_y;
            double* d_labels;
            hipMalloc((void**)&d_input, ((height * width) + 1) * sizeof(double));
            hipMalloc((void**)&d_weights, weights.size() * sizeof(double));
            hipMalloc((void**)&d_y, neuronas * sizeof(double));
            hipMalloc((void**)&d_labels, neuronas * sizeof(double));

            hipMemcpy(d_input, input.data(), ((height * width) + 1) * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_weights, weights.data(), weights.size() * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(d_labels, labels.data(), neuronas * sizeof(double), hipMemcpyHostToDevice);

            addKernel << <neuronas, (height * width) + 1 >> > (d_input, d_weights, d_y, height, width);
            hipDeviceSynchronize();
            hipMemcpy(y.data(), d_y, neuronas * sizeof(double), hipMemcpyDeviceToHost);

            bool converged = true;
            for (int i = 0; i < neuronas; i++) {
                if (y[i] != labels[i]) {
                        converged = false;
                        break;
                }
            }
            if (!converged)
                errors++;
                

            hipFree(d_weights);
            hipFree(d_input);
            hipFree(d_y);
            hipFree(d_labels);
            cout << ": " << (double)k / (double)nTraining * 100 << "%" << endl;
        }
        //cantidad--;
    //}

        cout << "Cantidad de errores: " << errors<<endl;

    return 0;
}
